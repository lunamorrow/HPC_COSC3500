#include "hip/hip_runtime.h"
#include "matrixMultiplyGPU.cuh"

__host__ void matrixMultiply_GPU(int N, const floatTypeCUDA* A, const floatTypeCUDA* B, floatTypeCUDA* C, int* flags, int flagCount)
{

    // this is the CPU, it hosts the GPU device -> only calls CPU functions
    // call the Kermal function!
    // declare the number of blocks per grid and the number of threads per block
    // int num_b = N/32; //((N+3)/4 + 32 - 1);
    dim3 num_blocks = dim3(64, 64, 1);
    dim3 num_threads = dim3(32, 32, 1);
    
    const size_t sz = N * N * sizeof(floatTypeCUDA);
    hipMemset(C, 0, sz);
    
    matrixMultiplyKernel_GPU<<<num_blocks, num_threads, sizeof(floatTypeCUDA)*1024>>>(N, A, B, C, 0, 0, 0);
}


__global__ void matrixMultiplyKernel_GPU(int N, const floatTypeCUDA* A, const floatTypeCUDA* B, floatTypeCUDA* C, int flag0, int flag1, int flag2)
{
    int tid = threadIdx.x * blockDim.y + threadIdx.y;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ double smem[];
    
    smem[tid] = __uint2double_rn(0);
        for (int k = 0; k < N; k+=16){
	    smem[tid] += ((B[col * N + k].x * A[k * N + row].x) +
	            (B[col * N + k + 1].x * A[(k+1) * N + row].x) +
	    (B[col * N + k+2].x * A[(k+2) * N + row].x) +
	    (B[col * N + k+3].x * A[(k+3) * N + row].x) +
	    (B[col * N + k+4].x * A[(k+4) * N + row].x) +
	    (B[col * N + k+5].x * A[(k+5) * N + row].x) +
	    (B[col * N + k+6].x * A[(k+6) * N + row].x) +
	    (B[col * N + k+7].x * A[(k+7) * N + row].x) +
	    (B[col * N + k+8].x * A[(k+8) * N + row].x) +
	    (B[col * N + k+9].x * A[(k+9) * N + row].x) +
	    (B[col * N + k+10].x * A[(k+10) * N + row].x) +
	    (B[col * N + k+11].x * A[(k+11) * N + row].x) +
	    (B[col * N + k+12].x * A[(k+12) * N + row].x) +
	    (B[col * N + k+13].x * A[(k+13) * N + row].x) +
	    (B[col * N + k+14].x * A[(k+14) * N + row].x) +
	    (B[col * N + k+15].x * A[(k+15) * N + row].x));


        }
    
    C[col * N + row].x = smem[tid];

}
